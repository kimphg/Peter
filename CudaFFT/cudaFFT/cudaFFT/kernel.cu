#include "hip/hip_runtime.h"
﻿//setx -m OPENCV_DIR D:\OpenCV\OpenCV331\opencv\build
//setx path "%path%;D:\OpenCV\OpenCV331\opencv\build\bin\Release\"
#include ""
//#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <winsock2.h>
#include <windows.h>
#include <conio.h>
#include <tchar.h>
#define HAVE_REMOTE// for pcap
#include "pcap.h"
#define HR2D_PK//
#define FRAME_LEN 2048
#define OUTPUT_FRAME_SIZE FRAME_LEN*2+FRAME_HEADER_SIZE
#define FFT_SIZE_MAX 256
#define BANG_KHONG 0
int mFFTSize = 32;
#define FFT_STEP (mFFTSize / 4)

#define MAX_IREC 2400
#pragma comment(lib, "user32.lib")
#pragma comment (lib, "Ws2_32.lib")
//file mapping
#define FRAME_HEADER_SIZE 34
using namespace std;
bool isPaused = false;
#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdio.h>
hipfftComplex *ramSignalTL;
hipfftComplex ramSignalNen[MAX_IREC][FRAME_LEN];
hipfftComplex ramImage[FRAME_LEN];
__global__ void complexMulKernel(hipfftComplex *res, const hipfftComplex *v1, const hipfftComplex *v2)
{
	int i = threadIdx.x;
	res[i].x = (v1[i].x * v2[i].x + v1[i].y * (v2[i].y));
	res[i].y = (v1[i].x * (-v2[i].y) + v1[i].y * v2[i].x);
}
class coreFFT
{
public:

	bool isActive;
	hipfftHandle planTL;
	//hipfftHandle planNenTH;
	//hipfftHandle planImageFFT;
	hipfftComplex *dSignalTL;
	//hipfftComplex *dSignalNenRes;
	//hipfftComplex *dSignalNen;
	//hipfftComplex *dImageNen;
	int mMemSizeTL;
	int mMemSizeNen;
	int mMemSizeImage;
	int mTichLuySize;//16
	int mFrameLen;
	coreFFT(int frameLen, int ntichluy)
	{
		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			isActive = false;
			return;
		}
		else
		{
			printf("\ncudaSetDevice on ");
			printf("\ncuda fft size:%d", ntichluy);
			printf("\nFFT ratio:1/%d", FFT_STEP);
		}
		mFrameLen = frameLen;
		mTichLuySize = ntichluy;
		mMemSizeTL = sizeof(hipfftComplex)* mTichLuySize*frameLen;
		if (hipfftPlan1d(&planTL, mTichLuySize, HIPFFT_C2C, frameLen) != HIPFFT_SUCCESS)
		{
			printf("\nFFT planTL failed to init");
			isActive = false;
			return;
		}
		ramSignalTL = new hipfftComplex[FRAME_LEN*mTichLuySize];
		// Allocate device memory for signal tich luy
		hipMalloc((void **)&dSignalTL, mMemSizeTL);
		
	}
	void exeFFTTL(hipfftComplex *h_signal)
	{
		hipMemcpy(dSignalTL, h_signal, mMemSizeTL, hipMemcpyHostToDevice);
		hipfftExecC2C(planTL, dSignalTL, dSignalTL, HIPFFT_FORWARD);

		if (hipGetLastError() != hipSuccess) {
			fprintf(stderr, "FFT kernel launch failed: %s\n", hipGetErrorString(hipGetLastError()));
			return;
		}
		hipMemcpy(h_signal, dSignalTL, mMemSizeTL, hipMemcpyDeviceToHost);
	}
	/*void exeFFTNen(hipfftComplex *h_signal, hipfftComplex* h_image)
	{
	//move signal to gpu and process fft forward
	hipMemcpy(dSignalNen, h_signal, mMemSizeNen, hipMemcpyHostToDevice);
	hipfftExecC2C(planNenTH, dSignalNen, dSignalNen, HIPFFT_FORWARD);
	//move image to gpu and process fft forward
	hipMemcpy(dImageNen, h_image, mMemSizeNen, hipMemcpyHostToDevice);
	hipfftExecC2C(planNenTH, dImageNen, dImageNen, HIPFFT_FORWARD);
	// Element wise complext multiplication
	for (int i = 0; i < mFrameLen; i += 1024)
	{
	complexMulKernel << <1, 1024 >> >(dSignalNenRes+i, dSignalNen+i, dImageNen+i);
	}

	hipfftExecC2C(planNenTH, dSignalNenRes, dSignalNenRes, HIPFFT_FORWARD);

	hipMemcpy(h_signal, dSignalNenRes, mMemSizeNen, hipMemcpyDeviceToHost);
	}*/
	~coreFFT()
	{
		delete[] ramSignalTL;
		hipfftDestroy(planTL);
		//hipfftDestroy(planNenTH);
		// cleanup memory
		hipFree(dSignalTL);
		printf("\nmemory clear");
		//hipFree(dSignalNen);
		//hipFree(dImageNen);
	}
};
//_______________________________________________________________________

struct DataFrame// buffer for data frame
{
	char header[FRAME_HEADER_SIZE];
	char dataPM_I[FRAME_LEN];
	char dataPM_Q[FRAME_LEN];
	short dataLen;
	bool isToFFT;
} dataBuff[MAX_IREC];
//unsigned int gyroValue = 0;

u_char outputFrame[OUTPUT_FRAME_SIZE];

int iProcessing = 0, iReady = 50;
void packet_handler(u_char *param, const struct pcap_pkthdr *header, const u_char *pkt_data);
void pcapRun();


int mSocket;
struct sockaddr_in si_peter;
struct sockaddr_in si_capin;
void socketInit()
{
	WSADATA wsa;
	//Initialise winsock
	printf("\nInitialising Winsock...");
	if (WSAStartup(MAKEWORD(2, 2), &wsa) != 0)
	{
		printf("Failed. Error Code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}
	printf("Initialised.\n");
	//init socket for UDP connect to Peter
	mSocket = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP);

	if (mSocket == SOCKET_ERROR)
	{
		printf("socket() failed with error code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}

	memset((char *)&si_capin, 0, sizeof(si_capin));
	si_capin.sin_family = AF_INET;
	si_capin.sin_port = htons(34566);//port "127.0.0.1"
	si_capin.sin_addr.S_un.S_addr = inet_addr("127.0.0.1");
	int ret = bind(mSocket, (struct sockaddr *)&si_capin, sizeof(struct sockaddr));
	if (ret == -1)
	{
		printf("Port busy");
		exit(EXIT_FAILURE);
	}
	//setup address structure
	memset((char *)&si_peter, 0, sizeof(si_peter));
	si_peter.sin_family = AF_INET;
	si_peter.sin_port = htons(31000);//port "127.0.0.1"
	si_peter.sin_addr.S_un.S_addr = inet_addr("127.0.0.1");

}
void socketDelete()
{
	closesocket(mSocket);
	WSACleanup();
}
void ProcessFrame(unsigned char*data, int len);

DWORD WINAPI ProcessDataBuffer(LPVOID lpParam);
DWORD WINAPI ProcessCommandBuffer(LPVOID lpParam);
void StartProcessing()
{
	CreateThread(
		NULL,                   // default security attributes
		0,                      // use default stack size  
		ProcessDataBuffer,       // thread function name
		NULL,          // argument to thread function 
		0,                      // use default creation flags 
		NULL);   // returns the thread identifier 
	CreateThread(
		NULL,                   // default security attributes
		0,                      // use default stack size  
		ProcessCommandBuffer,       // thread function name
		NULL,          // argument to thread function 
		0,                      // use default creation flags 
		NULL);   // returns the thread identifier 

}
coreFFT *mFFT;
FILE* pFile;
unsigned char buff[3000];

void ReplayData(const char* fileName)
{
	//char* mfileName = "C:\\Users\\Phuong-T1600\\Documents\\GitHub\\Peter\\VS\\x64\\Release\\raw_data_record_1538999224.dat";
	pFile = fopen(fileName, "rb");
	if (!pFile)
	{
		printf("\nfopen failed");
		return;
	}
	unsigned char len1, len2;
	unsigned long long dataSize = 0;
	while (!feof(pFile))
	{

		fread(&len1, 1, 1, pFile);
		fread(&len2, 1, 1, pFile);
		int len = (len1 << 8) | len2;
		printf("\ndatalen:%d", len);
		if (len > 3000)
		{
			printf("\nwrong datalen");
			break;
		}
		else if (len < 200)continue;
		fread(buff, 1, len, pFile);
		dataSize += len;
		ProcessFrame(buff, len);
		Sleep(5);
	}
	printf("\ntotal data sent:%d", dataSize);
}

int main(int argc, char **argv)
{

	/* start the capture */
	socketInit();
	mFFT = new coreFFT(FRAME_LEN, mFFTSize);
	StartProcessing();
	if (argc > 1)
	{
		char *fileName = argv[0];
		printf("\nreplay data file:");
		printf(fileName);
		ReplayData(fileName);
	}
	else pcapRun();

	return 0;
}
//precompiling code for FFT

// Complex data type


void pcapRun()
{
	pcap_if_t *alldevs;
	pcap_if_t *d;
	pcap_t *adhandle;
	char errbuf[PCAP_ERRBUF_SIZE];
	//
	/* Retrieve the device list on the local machine */
	if (pcap_findalldevs_ex((char*)PCAP_SRC_IF_STRING, NULL, &alldevs, errbuf) == -1)
	{
		//isRunning = false;
		printf(errbuf); return;
	}
	//isRunning = true;
	int i = 0;
	/* Print the list */
	for (d = alldevs; d; d = d->next)
	{
		printf("\n%d. %s", ++i, d->name);
		if (d->description)
			printf(" (%s)", d->description);
		else
			printf(" (No description available)");
	}
	d = alldevs;
	if ((adhandle = pcap_open(d->name,          // name of the device
		65536,            // portion of the packet to capture
		// 65536 guarantees that the whole packet will be captured on all the link layers
		PCAP_OPENFLAG_PROMISCUOUS,    // promiscuous mode
		1000,             // read timeout
		NULL,             // authentication on the remote machine
		errbuf            // error buffer
		)) == NULL)
	{
		/* Free the device list */
		pcap_freealldevs(alldevs);
		return;
	}
	printf("\nlistening on %s...\n", d->description);
	pcap_loop(adhandle, 0, packet_handler, NULL);
}
u_char dataOut[FRAME_LEN];
long int nFrames = 0;


char recvDatagram[1000];
DWORD WINAPI ProcessCommandBuffer(LPVOID lpParam)
{
	while (false)
	{
		int PeterAddrSize = sizeof (si_peter);
		int iResult = recvfrom(mSocket, recvDatagram, 1000, 0, (struct sockaddr *) &si_peter, &PeterAddrSize);
		if (iResult == SOCKET_ERROR) {
			wprintf(L"recvfrom failed with error %d\n", WSAGetLastError());
		}
	}
	return 0;
}
/*
int datatestI[MAX_IREC];
int datatestQ[MAX_IREC];
int datatestA[MAX_IREC];*/

DWORD WINAPI ProcessDataBuffer(LPVOID lpParam)
{


	while (true)
	{
		Sleep(1);
		while (iProcessing != iReady)
		{


			for (int ir = 0; ir < FRAME_LEN; ir++)
			{

				//ramSignalNen[iProcessing][ir].x = sqrt(double(dataBuff[iProcessing].dataPM_I[ir] * dataBuff[iProcessing].dataPM_I[ir] + dataBuff[iProcessing].dataPM_Q[ir] * dataBuff[iProcessing].dataPM_Q[ir]));//int(dataBuff[iProcessing].dataPM_I[ir]);
				ramSignalNen[iProcessing][ir].x = float(dataBuff[iProcessing].dataPM_I[ir]);
				ramSignalNen[iProcessing][ir].y = float(dataBuff[iProcessing].dataPM_Q[ir]);//0;// 
				//ramSignalNen[iProcessing][ir].y = 0;
			}
			if (!dataBuff[iProcessing].isToFFT || isPaused)
			{
				//jump to next period
				iProcessing++;
				if (iProcessing >= MAX_IREC)iProcessing = 0;
				continue;
			}
			int ia;
			for (int ir = 0; ir < FRAME_LEN; ir++)
			{
				ia = iProcessing;
				for (int i = 0; i < mFFTSize; i++)
				{
					ramSignalTL[ir*mFFTSize + i] = ramSignalNen[ia][ir];
					ia--;
					if (ia < 0)ia += MAX_IREC;
				}
			}
			// perform fft
			if (mFFT->isActive)mFFT->exeFFTTL((hipfftComplex*)ramSignalTL);
			//dataBuff[iProcessing].header[32] = gyroValue >> 8;
			//dataBuff[iProcessing].header[33] = gyroValue;

			memcpy(outputFrame, dataBuff[iProcessing].header, FRAME_HEADER_SIZE);

			for (int i = 0; i < FRAME_LEN; i++)
			{
				float maxAmp = 0;
				int indexMaxFFT = 0;
				//for (int j = 0; j<FFT_SIZE_MAX; j++)
				int fftSkip = BANG_KHONG*mFFTSize / 16;
				for (int j = fftSkip; j < mFFTSize - fftSkip; j++)
				{
					float ampl = (ramSignalTL[i*mFFTSize + j].x * ramSignalTL[i*mFFTSize + j].x) + (ramSignalTL[i*mFFTSize + j].y * ramSignalTL[i*mFFTSize + j].y);
					if (ampl>maxAmp)
					{
						maxAmp = ampl;
						indexMaxFFT = j;
					}
				}
				float res = sqrt(double(maxAmp) / double(mFFTSize));
				if (res > 255)res = 255;
				outputFrame[i + FRAME_HEADER_SIZE] = res;// u_char(sqrt(float(maxAmp)) / float(FFT_SIZE_MAX));
				outputFrame[i + FRAME_LEN + FRAME_HEADER_SIZE] = u_char(indexMaxFFT*16.0 / (mFFTSize));
			}
			sendto(mSocket, (char*)outputFrame, OUTPUT_FRAME_SIZE, 0, (struct sockaddr *) &si_peter, sizeof(si_peter));
		}
		//jump to next period
		iProcessing++;
		if (iProcessing >= MAX_IREC)iProcessing = 0;


	}



}

#define UDP_HEADER_LEN 42
void packet_handler(u_char *param, const struct pcap_pkthdr *pkt_header, const u_char *pkt_data)
{
	//    struct tm ltime;
	//    char timestr[16];
	//    time_t local_tv_sec;

	/*
	* unused variables
	*/
	//    (VOID)(param);
	//    (VOID)(pkt_data);

	/* convert the timestamp to readable format */
	//    local_tv_sec = header->ts.tv_sec;
	//    localtime_s(&ltime, &local_tv_sec);
	//    strftime( timestr, sizeof timestr, "%H:%M:%S", &ltime);

	if (pkt_header->len<200)return;
	int port = ((*(pkt_data + 36) << 8) | (*(pkt_data + 37)));
	if (port == 5000)
	{
		/*
		+ 0: 1024 byte đầu kênh I
		+ 1: 1024 byte sau kênh I
		+ 2: 1024 byte đầu kênh Q
		+ 3: 1024 byte sau kênh Q
		+ 4: 256 byte máy hỏi
		+ 5: 1024 byte tín hiệu giả L/tục (512 byte đầu là I, 512 byte sau là Q)
		+ 6: 1024 byte sau kênh I tín hiệu xung đơn
		+ 7: 1024 byte sau kênh Q tín hiệu xung đơn

		*/
		u_char* data = (u_char*)pkt_data + UDP_HEADER_LEN;
		ProcessFrame(data, pkt_header->len);

	}


}
/*void packet_handler_compress(u_char *param, const struct pcap_pkthdr *pkt_header, const u_char *pkt_data)
{
if (pkt_header->len<1000)return;
if (((*(pkt_data + 36) << 8) | (*(pkt_data + 37))) != 5000)
{
return;
}
u_char* data = (u_char*)pkt_data + UDP_HEADER_LEN;
if (data[0] == 0)		//I chanel first part
{
iReady++;
if (iReady >= MAX_IREC)iReady = 0;
memcpy(dataBuff[iReady].header, data, FRAME_HEADER_SIZE);
memcpy(dataBuff[iReady].dataPM_I, data + FRAME_HEADER_SIZE, 1024);
}
else if (data[0] == 2) //Q chanel first part
{
memcpy(dataBuff[iReady].dataPM_Q, data + FRAME_HEADER_SIZE, 1024);
}
else if (data[0] == 1) //I chanel second part
{
memcpy(dataBuff[iReady].dataPM_I + 1024, data + FRAME_HEADER_SIZE, 1024);
}
else if (data[0] == 3) //Q chanel second part
{
memcpy(dataBuff[iReady].dataPM_Q + 1024, data + FRAME_HEADER_SIZE, 1024);
}
return;
}*/
/*
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   STT |   Byte    |   Chức                                              |
|       |           |   năng                                              |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   1   |   0       |   Id gói                                            |
|       |           |   tin:                                              |
|       |           |   0,1,2,3:                                          |
|       |           |   iq th mã pha (mỗi kênh 2048 byte)                 |
|       |           |   4: 256                                            |
|       |           |   byte máy hỏi, mỗi bít một o_cu_ly                 |
|       |           |   5: iq th                                          |
|       |           |   giả liên tục, 512 byte i, 512 byte q              |
|       |           |   6,7: iq                                           |
|       |           |   cho tín hiệu xung đơn, mỗi kênh 1024 byte         |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   2   |   1, 2, 3 |   Byte cho                                          |
|       |           |   báo hỏng:                                         |
|       |           |   1: loại                                           |
|       |           |   mô-đun, (0, 1, 2, 3)                              |
|       |           |   2: Loại                                           |
|       |           |   tham số (bb, cc, dd)                              |
|       |           |   3: Tham                                           |
|       |           |   số mô-đun                                         |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   3   |   4       |   Phân giải                                         |
|       |           |   ra đa: 0 (15m), 1 (30m)......                     |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   4   |   5,6     |   Loại tín                                          |
|       |           |   hiệu phát và tham số:                             |
|       |           |   5: loại                                           |
|       |           |   th phát (0: xung đơn; 1: mã pha; 2: giả ltuc)     |
|       |           |   6: tham                                           |
|       |           |   số cho loại th trên                               |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   5   |   7,8     |   Hai byte                                          |
|       |           |   trung bình tạp máy thu (ktra báo hỏng tuyến thu)  |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   6   |   9, 10,  |   4 byte                                            |
|       |   11, 12  |   quay an-ten                                       |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   7   |   13, 14  |   Hai byte                                          |
|       |           |   hướng tàu                                         |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   8   |   15, 16  |   Hai byte                                          |
|       |           |   hướng mũi tàu                                     |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   9   |   17, 18  |   Hai byte                                          |
|       |           |   tốc độ tàu                                        |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   10  |   19      |   Thông                                             |
|       |           |   báo chế độ chủ đông - bị động, tốc độ quay an-ten |
|       |           |   - bít thấp                                        |
|       |           |   thông báo cđ-bđ (1: chủ động)                     |
|       |           |   - 4 bít                                           |
|       |           |   cao là tốc độ an-ten                              |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   11  |   20      |   Thông                                             |
|       |           |   báo tần số phát và đặt mức tín hiệu:              |
|       |           |   - 4 bít                                           |
|       |           |   cuối là tần số phát                               |
|       |           |   - 4 bít                                           |
|       |           |   cao là đặt mức th                                 |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   12  |   21      |   Thông                                             |
|       |           |   báo chọn thang cự ly và bật/tắt AM2:              |
|       |           |   - 4 bít                                           |
|       |           |   cuối là thang cự ly (0: 2 lý; 1: 4 lý.....)       |
|       |           |   - 4 bít                                           |
|       |           |   cao là báo bật/tắt AM2: 0: tắt, 1: bật            |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
|       |           |                                                     |
|   13  |   22      |   Thông                                             |
|       |           |   báo số điểm FFT:                                  |
|       |           |   1(fft8);                                          |
|       |           |   2(fft16) ;...;32(fft256)                          |
|       |           |                                                     |
+-------+-----------+-----------------------------------------------------+
Id gói                                            |
|       |           |   tin:                                              |
|       |           |   0,1,2,3:                                          |
|       |           |   iq th mã pha (mỗi kênh 2048 byte)                 |
|       |           |   4: 256                                            |
|       |           |   byte máy hỏi, mỗi bít một o_cu_ly                 |
|       |           |   5: iq th                                          |
|       |           |   giả liên tục, 512 byte i, 512 byte q              |
|       |           |   6,7: iq                                           |
|       |           |   cho tín hiệu xung đơn, mỗi kênh 1024 byte
*/
static int fftID = -1;
void ProcessFrame(unsigned char*data, int len)
{
	int iNext = iReady + 1;
	if (iNext >= MAX_IREC)iNext = 0;
	int newfftID = data[22];
	if(fftID!=newfftID)
	{
		fftID = newfftID;
		mFFTSize = pow(2.0, fftID + 2);
		if (mFFTSize > 512 || mFFTSize < 4)mFFTSize = 32;
		isPaused = true;
		Sleep(20);
		//if (mFFT)delete mFFT;
		mFFT = new coreFFT(FRAME_LEN, mFFTSize);
		Sleep(5);
		isPaused = false;
	}
	memcpy(dataBuff[iNext].header, data, FRAME_HEADER_SIZE);
	
	bool isLastFrame = false;
	if (data[0] == 0)		//0: 1024 byte đầu kênh I
	{
		//memcpy(dataBuff[iNext].header, data, FRAME_HEADER_SIZE);
		memcpy(dataBuff[iNext].dataPM_I, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 1) //1: 1024 byte sau kênh I
	{
		memcpy(dataBuff[iNext].dataPM_I + 1024, data + FRAME_HEADER_SIZE, 1024);

	}
	else if (data[0] == 2) //2: 1024 byte đầu kênh Q
	{
		memcpy(dataBuff[iNext].dataPM_Q, data + FRAME_HEADER_SIZE, 1024);

	}
	else if (data[0] == 3) //3: 1024 byte sau kênh Q
	{
		memcpy(dataBuff[iNext].dataPM_Q + 1024, data + FRAME_HEADER_SIZE, 1024);
		dataBuff[iNext].dataLen = FRAME_LEN;
		isLastFrame = true;

	}
	else if (data[0] == 4) //4: máy hỏi
	{
		
		sendto(mSocket, (char*)data, len, 0, (struct sockaddr *) &si_peter, sizeof(si_peter));
		//isLastFrame = true;

	}
	else if (data[0] == 5) //5: 1024 byte tín hiệu giả L/tục (512 byte đầu là I, 512 byte sau là Q) 
	{
		memcpy(dataBuff[iNext].dataPM_I, data + FRAME_HEADER_SIZE, 512);
		memcpy(dataBuff[iNext].dataPM_Q, data + FRAME_HEADER_SIZE + 512, 512);
		dataBuff[iNext].dataLen = 512;
		isLastFrame = true;
	}
	else if (data[0] == 6) //6: 1024 byte sau kênh I tín hiệu xung đơn 
	{
		memcpy(dataBuff[iNext].dataPM_I, data + FRAME_HEADER_SIZE, 1024);
		dataBuff[iNext].dataLen = 1024;

	}
	else if (data[0] == 7) //7: 1024 byte sau kênh Q tín hiệu xung đơn 
	{
		memcpy(dataBuff[iNext].dataPM_Q, data + FRAME_HEADER_SIZE, 1024);
		dataBuff[iNext].dataLen = 1024;
		isLastFrame = true;

	}
	if (isLastFrame)
	{
		iReady++;
		dataBuff[iNext].isToFFT = ((iNext%FFT_STEP) == 0);
		if (iReady >= MAX_IREC)iReady = 0;
	}
	return;

}