#include "hip/hip_runtime.h"
//setx -m OPENCV_DIR D:\OpenCV\OpenCV331\opencv\build
//setx path "%path%;D:\OpenCV\OpenCV331\opencv\build\bin\Release\"
#include ""
//#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <winsock2.h>
#include <windows.h>
#include <conio.h>
#include <tchar.h>
#define HAVE_REMOTE// for pcap
#include "pcap.h"

#define FRAME_LEN 2048
#define MAX_IREC 4000
#pragma comment(lib, "user32.lib")
#pragma comment (lib, "Ws2_32.lib")
//file mapping
#define BUF_SIZE 256
#define FRAME_HEADER_SIZE 34

//using namespace cv;
using namespace std;
//#include "FFTcore.cuh"
// includes for FFT
#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdio.h>

__global__ void complexMulKernel(hipfftComplex *res, const hipfftComplex *v1, const hipfftComplex *v2)
{
	int i = threadIdx.x;
	res[i].x = (v1[i].x * v2[i].x + v1[i].y * (v2[i].y)) / 2048.0 / 2048.0;
	res[i].y = (v1[i].x * (-v2[i].y) + v1[i].y * v2[i].x) / 2048.0 / 2048.0;
}
class coreFFT
{
public:


	hipfftHandle planTL;
	hipfftHandle planNenTH;
	//hipfftHandle planImageFFT;
	hipfftComplex *dSignalTL;
	hipfftComplex *dSignalNenRes;
	hipfftComplex *dSignalNen;
	hipfftComplex *dImageNen;
	int mMemSizeTL;
	int mMemSizeNen;
	int mMemSizeImage;
	int mTichLuySize;//16
	int mFrameLen;
	coreFFT(int frameLen, int ntichluy)
	{
		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			
		}
		mFrameLen = frameLen;
		mTichLuySize = ntichluy;
		mMemSizeTL = sizeof(hipfftComplex)* mTichLuySize*frameLen;
		mMemSizeNen = sizeof(hipfftComplex)*frameLen;
		mMemSizeImage = sizeof(hipfftComplex)*frameLen;


		if (hipfftPlan1d(&planTL, mTichLuySize, HIPFFT_C2C, frameLen) != HIPFFT_SUCCESS)printf("\nFFT planTL failed to init");
		// Allocate device memory for signal tich luy
		hipMalloc((void **)&dSignalTL, mMemSizeTL);

		if (hipfftPlan1d(&planNenTH, frameLen, HIPFFT_C2C, 1) != HIPFFT_SUCCESS)printf("\nFFT planTL failed to init");
		// Allocate device memory for signal nen
		hipMalloc((void **)&dSignalNen, mMemSizeNen);
		hipMalloc((void **)&dSignalNenRes, mMemSizeNen);

		//if (hipfftPlan1d(&planImageFFT, frameLen, HIPFFT_C2C, 1) != HIPFFT_SUCCESS)printf("\nFFT planTL failed to init");
		// Allocate device memory for image nen
		hipMalloc((void **)&dImageNen, mMemSizeNen);
	}
	void exeFFTTL(hipfftComplex *h_signal)
	{
		hipMemcpy(dSignalTL, h_signal, mMemSizeTL, hipMemcpyHostToDevice);
		hipfftExecC2C(planTL, dSignalTL, dSignalTL, HIPFFT_FORWARD);
		hipMemcpy(h_signal, dSignalTL, mMemSizeTL, hipMemcpyDeviceToHost);
	}
	void exeFFTNen(hipfftComplex *h_signal, hipfftComplex* h_image)
	{
		//move signal to gpu and process fft forward
		hipMemcpy(dSignalNen, h_signal, mMemSizeNen, hipMemcpyHostToDevice);
		hipfftExecC2C(planNenTH, dSignalNen, dSignalNen, HIPFFT_FORWARD);
		//move image to gpu and process fft forward
		hipMemcpy(dImageNen, h_image, mMemSizeNen, hipMemcpyHostToDevice);
		hipfftExecC2C(planNenTH, dImageNen, dImageNen, HIPFFT_FORWARD);
		// Element wise complext multiplication
		for (int i = 0; i < mFrameLen; i += 1024)
		{
			complexMulKernel << <1, 1024 >> >(dSignalNenRes+i, dSignalNen+i, dImageNen+i);
		}
		
		/*hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			
		}*/

		/*
		hipMemcpy(h_image, dImageNen, mMemSizeNen, hipMemcpyDeviceToHost);
		hipMemcpy(h_signal, dSignalNen, mMemSizeNen, hipMemcpyDeviceToHost);
		for (int i = 0; i < mFrameLen; i++)
		{
			float x = h_image[i].x * h_signal[i].x + h_image[i].y * h_signal[i].y;
			float y = h_signal[i].x * (-h_image[i].y) + h_signal[i].y * h_image[i].x;
			h_signal[i].x = x/1000000.0;
			h_signal[i].y = y/1000000.0;
		}
		hipMemcpy(dSignalNenRes, h_signal, mMemSizeNen, hipMemcpyHostToDevice);
		*/
		hipfftExecC2C(planNenTH, dSignalNenRes, dSignalNenRes, HIPFFT_FORWARD);
		
		hipMemcpy(h_signal, dSignalNenRes, mMemSizeNen, hipMemcpyDeviceToHost);
	}
	~coreFFT()
	{
		hipfftDestroy(planTL);
		hipfftDestroy(planNenTH);
		// cleanup memory
		hipFree(dSignalTL);
		hipFree(dSignalNen);
		hipFree(dImageNen);
	}
};
//_______________________________________________________________________

struct DataFrame// buffer for data frame
{
	char header[FRAME_HEADER_SIZE];
	char dataI[FRAME_LEN];
	char dataQ[FRAME_LEN];
	char image256[256];
	bool isToFFT;
} dataBuff[MAX_IREC];

#define OUTPUT_FRAME_SIZE FRAME_LEN*2+FRAME_HEADER_SIZE

u_char outputFrame[OUTPUT_FRAME_SIZE];

int iProcessing=0,iReady = 5;
void packet_handler(u_char *param, const struct pcap_pkthdr *header, const u_char *pkt_data);
void pcapRun();


int mSocket;
struct sockaddr_in si_other;
void socketInit()
{
	WSADATA wsa;
	//Initialise winsock
	printf("\nInitialising Winsock...");
	if (WSAStartup(MAKEWORD(2, 2), &wsa) != 0)
	{
		printf("Failed. Error Code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}
	printf("Initialised.\n");
	//create socket
	mSocket = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP);
	if (mSocket == SOCKET_ERROR)
	{
		printf("socket() failed with error code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}
	//setup address structure
	memset((char *)&si_other, 0, sizeof(si_other));
	si_other.sin_family = AF_INET;
	si_other.sin_port = htons(34567);//port "127.0.0.1"
	si_other.sin_addr.S_un.S_addr = inet_addr("127.0.0.1");

}
void socketDelete()
{
	closesocket(mSocket);
	WSACleanup();
}

DWORD WINAPI ProcessDataBuffer(LPVOID lpParam);
void StartProcessing()
{
	CreateThread(
		NULL,                   // default security attributes
		0,                      // use default stack size  
		ProcessDataBuffer,       // thread function name
		NULL,          // argument to thread function 
		0,                      // use default creation flags 
		NULL);   // returns the thread identifier 

}
coreFFT *mFFT;
#define FFT_SIZE 16
int mFFTSkip = 4;

int main()
{

	/* start the capture */
	socketInit();
	mFFT = new coreFFT(FRAME_LEN, FFT_SIZE);
	StartProcessing();
	pcapRun();
	
    return 0;
}
//precompiling code for FFT

// Complex data type


void pcapRun()
{
	pcap_if_t *alldevs;
	pcap_if_t *d;
	pcap_t *adhandle;
	char errbuf[PCAP_ERRBUF_SIZE];
	//
	/* Retrieve the device list on the local machine */
	if (pcap_findalldevs_ex((char*)PCAP_SRC_IF_STRING, NULL, &alldevs, errbuf) == -1)
	{
		//isRunning = false;
		printf(errbuf); return;
	}
	//isRunning = true;
	int i = 0;
	/* Print the list */
	for (d = alldevs; d; d = d->next)
	{
		printf("%d. %s", ++i, d->name);
		if (d->description)
			printf(" (%s)\n", d->description);
		else
			printf(" (No description available)\n");
	}
	d = alldevs;
	if ((adhandle = pcap_open(d->name,          // name of the device
		65536,            // portion of the packet to capture
		// 65536 guarantees that the whole packet will be captured on all the link layers
		PCAP_OPENFLAG_PROMISCUOUS,    // promiscuous mode
		1000,             // read timeout
		NULL,             // authentication on the remote machine
		errbuf            // error buffer
		)) == NULL)
	{
		/* Free the device list */
		pcap_freealldevs(alldevs);
		return;
	}
	printf("\nlistening on %s...\n", d->description);
	pcap_loop(adhandle, 0, packet_handler, NULL);
}
u_char dataOut[FRAME_LEN];
long int nFrames = 0;

hipfftComplex ramSignalTL[FRAME_LEN][FFT_SIZE];
hipfftComplex ramSignalNen[MAX_IREC][FRAME_LEN];
hipfftComplex ramImage[FRAME_LEN];

DWORD WINAPI ProcessDataBuffer(LPVOID lpParam)
{
	while (true)
	{

		Sleep(1);
		while (iProcessing!= iReady )
		{
			iProcessing++;
			if (iProcessing >= MAX_IREC)iProcessing = 0;
			for (int ir = 0; ir < FRAME_LEN; ir++)
			{
				ramSignalNen[iProcessing][ir].x = int(dataBuff[iProcessing].dataI[ir]);
				ramSignalNen[iProcessing][ir].y = int(dataBuff[iProcessing].dataQ[ir]);
			}
			for (int ir = 0; ir < FRAME_LEN; ir++)
			{
				if (ir < 256)
				{
					ramImage[ir].x = ramSignalNen[iProcessing][ir+500].x;//dataBuff[iProcessing].image256[ir];
				}
				else
				{
					ramImage[ir].x = 0;
				}
				ramImage[ir].y = 0;
			}
			//bat dau loc nen anh guong
			mFFT->exeFFTNen(ramSignalNen[iProcessing], ramImage);
			/*int sum = 0;
			for (int i = 0; i < FRAME_LEN; i++)
			{
				int outx =  int(sqrt((ramSignalNen[iProcessing][i].x)*(ramSignalNen[iProcessing][i].x) + (ramSignalNen[iProcessing][i].y)*(ramSignalNen[iProcessing][i].y)));
				sum += outx;
				if (outx>255)
					outx = 255;
				//bien bo
				outputFrame[i + FRAME_HEADER_SIZE] = outx;// u_char((ramSignalNen[iProcessing][i].x)*(ramSignalNen[iProcessing][i].x) + (ramSignalNen[iProcessing][i].y)*(ramSignalNen[iProcessing][i].y));
				//kenh dopler
				outputFrame[i + FRAME_LEN + FRAME_HEADER_SIZE] = 0;
			}
			if (sum / 2048.0 > 200)
			{
				sum = sum;
			}*/
			//memcpy((char*)outputFrame + FRAME_HEADER_SIZE, ramSignalNen[iProcessing],1024);
			//memcpy((char*)outputFrame + FRAME_HEADER_SIZE + 1024, ramSignalNen[iProcessing]+1024, 1024);
			//memcpy(outputFrame, dataBuff[iProcessing].header, FRAME_HEADER_SIZE);
			//sendto(mSocket, (char*)outputFrame, OUTPUT_FRAME_SIZE, 0, (struct sockaddr *) &si_other, sizeof(si_other));
			//
			//continue;
			//tich luy fft
			if (!dataBuff[iProcessing].isToFFT)continue;

			if (iProcessing >= MAX_IREC)iProcessing -= MAX_IREC;
			nFrames++;
			for (int ir = 0; ir < FRAME_LEN; ir++)
			{
				int ia = iProcessing;
				for (int i = 0; i < FFT_SIZE; i++)
				{
					ramSignalTL[ir][i] = ramSignalNen[ia][ir];
					ia--;
					if (ia < 0)ia += MAX_IREC;
				}
			}
			
			/*if ((dataBuff[iProcessing].azi != oldAzi + 1) && (oldAzi!=2047))
			{
				printf("\nAzi:%d Count:%d", dataBuff[iProcessing].azi, nFrames);
				printf("Azi old:%d", oldAzi);
			}*/
			
			/*oldAzi = dataBuff[iProcessing].azi;
			*/
			//printf("\nAzi:%d Count:%d", dataBuff[iProcessing].azi, nFrames);
			int iDisplay = 477;

			if (nFrames < 50)
			{
				
				printf("\nInput I:");
				for (int i = 0; i < FFT_SIZE; i++)
				{
					printf("%3.2f ", ramSignalTL[iDisplay][i].x);
				}
				printf("\nInput Q:");
				for (int i = 0; i < FFT_SIZE; i++)
				{
					printf("%3.2f ", ramSignalTL[iDisplay][i].y);
				}
			}
			
			// perform fft

			mFFT->exeFFTTL((hipfftComplex*)ramSignalTL);
			
			memcpy(outputFrame, dataBuff[iProcessing].header, FRAME_HEADER_SIZE);
			
			for (int i = 0; i < FRAME_LEN; i++)
			{
				if (i == iDisplay)
				{
					i = i;
				}
				int maxAmp = 0, indexMaxFFT = 0;
				for (int j = 0; j<FFT_SIZE; j++)
				{
					int ampl = (ramSignalTL[i][j].x * ramSignalTL[i][j].x) + (ramSignalTL[i][j].y * ramSignalTL[i][j].y);
					if (ampl>maxAmp)
					{
						maxAmp = ampl;
						indexMaxFFT = j;
					}
				}
				outputFrame[i + FRAME_HEADER_SIZE] = u_char(sqrt(float(maxAmp)) / float(FFT_SIZE));
				outputFrame[i + FRAME_LEN + FRAME_HEADER_SIZE] = u_char(indexMaxFFT);
			}
			sendto(mSocket, (char*)outputFrame, OUTPUT_FRAME_SIZE, 0, (struct sockaddr *) &si_other, sizeof(si_other));
			//printf(" max FFT:%d",indexMaxFFT);
			
			if (nFrames < 50)
			{
				
				printf("\nFFT I:");
				for (int i = 0; i < FFT_SIZE; i++)
				{
					printf("%3.2f ", ramSignalTL[iDisplay][i].x);
				}
				printf("\nFFT Q:");
				for (int i = 0; i < FFT_SIZE; i++)
				{
					printf("%3.2f ", ramSignalTL[iDisplay][i].y);
				}
			}
			
			
		}
	}
	


}

#define UDP_HEADER_LEN 42
void packet_handler(u_char *param, const struct pcap_pkthdr *pkt_header, const u_char *pkt_data)
{
	//    struct tm ltime;
	//    char timestr[16];
	//    time_t local_tv_sec;

	/*
	* unused variables
	*/
	//    (VOID)(param);
	//    (VOID)(pkt_data);

	/* convert the timestamp to readable format */
	//    local_tv_sec = header->ts.tv_sec;
	//    localtime_s(&ltime, &local_tv_sec);
	//    strftime( timestr, sizeof timestr, "%H:%M:%S", &ltime);

	if (pkt_header->len<1000)return;
	//pkt_header->
	if (((*(pkt_data + 36) << 8) | (*(pkt_data + 37))) != 5000)
	{
		//printf("\nport:%d",((*(pkt_data+36)<<8)|(*(pkt_data+37))));
		return;
	}
	int iNext = iReady + 1;
	if (iNext >= MAX_IREC)iNext = 0;
	u_char* data = (u_char*)pkt_data + UDP_HEADER_LEN;
	if (data[0] == 0)		//I chanel first part
	{
		
		memcpy(dataBuff[iNext].header, data, FRAME_HEADER_SIZE);
		memcpy(dataBuff[iNext].dataI, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 2) //Q chanel first part
	{
		memcpy(dataBuff[iNext].dataQ, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 1) //I chanel second part
	{
		memcpy(dataBuff[iNext].dataI + 1024, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 3) //Q chanel second part
	{
		memcpy(dataBuff[iNext].dataQ + 1024, data + FRAME_HEADER_SIZE, 1024);
		dataBuff[iNext].isToFFT = ((iNext%mFFTSkip)==0);
		iReady++;
		if (iReady >= MAX_IREC)iReady = 0;
	}
	return;
}
void packet_handler_compress(u_char *param, const struct pcap_pkthdr *pkt_header, const u_char *pkt_data)
{
	if (pkt_header->len<1000)return;
	if (((*(pkt_data + 36) << 8) | (*(pkt_data + 37))) != 5000)
	{
		return;
	}
	u_char* data = (u_char*)pkt_data + UDP_HEADER_LEN;
	if (data[0] == 0)		//I chanel first part
	{
		iReady++;
		if (iReady >= MAX_IREC)iReady = 0;
		memcpy(dataBuff[iReady].header, data, FRAME_HEADER_SIZE);
		memcpy(dataBuff[iReady].dataI, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 2) //Q chanel first part
	{
		memcpy(dataBuff[iReady].dataQ, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 1) //I chanel second part
	{
		memcpy(dataBuff[iReady].dataI + 1024, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 3) //Q chanel second part
	{
		memcpy(dataBuff[iReady].dataQ + 1024, data + FRAME_HEADER_SIZE, 1024);
	}
	return;
}