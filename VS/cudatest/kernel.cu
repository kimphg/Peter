#include "hip/hip_runtime.h"
//setx -m OPENCV_DIR D:\OpenCV\OpenCV331\opencv\build
//setx path "%path%;D:\OpenCV\OpenCV331\opencv\build\bin\Release\"
#include "hip/hip_runtime.h"
#include ""
//#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <winsock2.h>
#include <windows.h>
#include <conio.h>
#include <tchar.h>
// includes for FFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#define FFT_SIZE 16
//#define 
#define FRAME_LEN 2048
#pragma comment(lib, "user32.lib")
#pragma comment (lib, "Ws2_32.lib")
//file mapping
#define BUF_SIZE 256
#define FRAME_HEADER_SIZE 34
TCHAR szName[] = TEXT("Global\\RadarData");
#define HAVE_REMOTE// for pcap
#include "pcap.h"
//using namespace cv;
using namespace std;
struct DataFrame// buffer for data frame
{
	char header[FRAME_HEADER_SIZE];
	char dataI[FRAME_LEN];
	char dataQ[FRAME_LEN];
};
#define MAX_IREC 10000
#define OUTPUT_FRAME_SIZE FRAME_LEN*2+FRAME_HEADER_SIZE
DataFrame dataBuff[MAX_IREC];
u_char outputFrame[OUTPUT_FRAME_SIZE];
hipfftHandle plan;
int iRead=0,iRec = 1;
void packet_handler(u_char *param, const struct pcap_pkthdr *header, const u_char *pkt_data);
void pcapRun();
hipfftComplex *d_signal;
int mem_size = sizeof(hipfftComplex)* FFT_SIZE*FRAME_LEN;
int mSocket;
struct sockaddr_in si_other;
void socketInit()
{
	WSADATA wsa;
	//Initialise winsock
	printf("\nInitialising Winsock...");
	if (WSAStartup(MAKEWORD(2, 2), &wsa) != 0)
	{
		printf("Failed. Error Code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}
	printf("Initialised.\n");
	//create socket
	mSocket = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP);
	if (mSocket == SOCKET_ERROR)
	{
		printf("socket() failed with error code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}
	//setup address structure
	memset((char *)&si_other, 0, sizeof(si_other));
	si_other.sin_family = AF_INET;
	si_other.sin_port = htons(34567);//port "127.0.0.1"
	si_other.sin_addr.S_un.S_addr = inet_addr("127.0.0.1");

}
void socketDelete()
{
	closesocket(mSocket);
	WSACleanup();
}
void cufftInit()
{
	if (hipfftPlan1d(&plan, FFT_SIZE, HIPFFT_C2C, FRAME_LEN) != HIPFFT_SUCCESS)printf("\nFFT plan failed to init");
	// Allocate device memory for signal
	hipMalloc((void **)&d_signal, mem_size);
}
void cufftExit()
{
	hipfftDestroy(plan);
	// cleanup memory
	hipFree(d_signal);
}
DWORD WINAPI ProcessDataBuffer(LPVOID lpParam);
void StartProcessing()
{
	CreateThread(
		NULL,                   // default security attributes
		0,                      // use default stack size  
		ProcessDataBuffer,       // thread function name
		NULL,          // argument to thread function 
		0,                      // use default creation flags 
		NULL);   // returns the thread identifier 

}
int main()
{

	/* start the capture */
	socketInit();
	cufftInit();
	StartProcessing();
	pcapRun();
	cufftExit();
    return 0;
}
//precompiling code for FFT

// Complex data type


void pcapRun()
{

	pcap_if_t *alldevs;
	pcap_if_t *d;
	pcap_t *adhandle;
	char errbuf[PCAP_ERRBUF_SIZE];
	//
	/* Retrieve the device list on the local machine */
	if (pcap_findalldevs_ex((char*)PCAP_SRC_IF_STRING, NULL, &alldevs, errbuf) == -1)
	{
		//isRunning = false;
		printf(errbuf); return;
	}
	//isRunning = true;
	int i = 0;
	/* Print the list */
	for (d = alldevs; d; d = d->next)
	{
		printf("%d. %s", ++i, d->name);
		if (d->description)
			printf(" (%s)\n", d->description);
		else
			printf(" (No description available)\n");
	}
	d = alldevs;
	if ((adhandle = pcap_open(d->name,          // name of the device
		65536,            // portion of the packet to capture
		// 65536 guarantees that the whole packet will be captured on all the link layers
		PCAP_OPENFLAG_PROMISCUOUS,    // promiscuous mode
		1000,             // read timeout
		NULL,             // authentication on the remote machine
		errbuf            // error buffer
		)) == NULL)
	{
		/* Free the device list */
		pcap_freealldevs(alldevs);
		return;
	}
	printf("\nlistening on %s...\n", d->description);
	pcap_loop(adhandle, 0, packet_handler, NULL);
}

void CalculateFFT(hipfftComplex *h_signal)
{
	hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice);
	hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);
	hipMemcpy(h_signal, d_signal, mem_size, hipMemcpyDeviceToHost);
}
long int nFrames = 0;

u_char dataOut[FRAME_LEN];
DWORD WINAPI ProcessDataBuffer(LPVOID lpParam)
{
	//int oldAzi;
	while (true)
	{
		Sleep(2);
		int iCur = (iRec>>2) - 1;
		if (iCur < 0)iCur += MAX_IREC>>2;
		
		while ((iRead>>2) != iCur )
		{
			iRead+=4;
			
			if (iRead >= MAX_IREC)iRead -= MAX_IREC;
			nFrames++;
			hipfftComplex h_signal[FRAME_LEN][FFT_SIZE];// (hipfftComplex *)malloc(sizeof(hipfftComplex)* FFT_SIZE);
			for (int ir = 0; ir < FRAME_LEN; ir++)
			{
				int ia = iRead;
				
				for (int i = 0; i < FFT_SIZE; i++)
				{
					h_signal[ir][i].x = dataBuff[ia].dataI[ir];
					h_signal[ir][i].y = dataBuff[ia].dataQ[ir];
					ia--;
					if (ia < 0)ia += MAX_IREC;
				}
			}
			
			
			/*if ((dataBuff[iRead].azi != oldAzi + 1) && (oldAzi!=2047))
			{
				printf("\nAzi:%d Count:%d", dataBuff[iRead].azi, nFrames);
				printf("Azi old:%d", oldAzi);
			}*/
			
			/*oldAzi = dataBuff[iRead].azi;
			*/
			//printf("\nAzi:%d Count:%d", dataBuff[iRead].azi, nFrames);
			int iDisplay = 477;
			if (nFrames < 50)
			{
				
				printf("\nInput I:");
				for (int i = 0; i < FFT_SIZE; i++)
				{
					printf("%3.2f ", h_signal[iDisplay][i].x);
				}
				printf("\nInput Q:");
				for (int i = 0; i < FFT_SIZE; i++)
				{
					printf("%3.2f ", h_signal[iDisplay][i].y);
				}
			}
			// perform fft
			CalculateFFT((hipfftComplex*)h_signal);
			memcpy(outputFrame, dataBuff[iRead].header, FRAME_HEADER_SIZE);
			
			for (int i = 0; i < FRAME_LEN; i++)
			{
				if (i == iDisplay)
				{
					i = i;
				}
				int maxAmp = 0, indexMaxFFT = 0;
				for (int j = 0; j<FFT_SIZE; j++)
				{
					int ampl = (h_signal[i][j].x * h_signal[i][j].x) + (h_signal[i][j].y * h_signal[i][j].y);
					if (ampl>maxAmp)
					{
						maxAmp = ampl;
						indexMaxFFT = j;
					}
				}
				outputFrame[i + FRAME_HEADER_SIZE] = u_char(sqrt(float(maxAmp)) / float(FFT_SIZE));
				outputFrame[i + FRAME_LEN + FRAME_HEADER_SIZE] = u_char(indexMaxFFT);
			}
			sendto(mSocket, (char*)outputFrame, OUTPUT_FRAME_SIZE, 0, (struct sockaddr *) &si_other, sizeof(si_other));
			//printf(" max FFT:%d",indexMaxFFT);
			
			if (nFrames < 50)
			{
				
				printf("\nFFT I:");
				for (int i = 0; i < FFT_SIZE; i++)
				{
					printf("%3.2f ", h_signal[iDisplay][i].x);
				}
				printf("\nFFT Q:");
				for (int i = 0; i < FFT_SIZE; i++)
				{
					printf("%3.2f ", h_signal[iDisplay][i].y);
				}
			}
			
			
		}
	}
	


}

#define UDP_HEADER_LEN 42
void packet_handler(u_char *param, const struct pcap_pkthdr *pkt_header, const u_char *pkt_data)
{
	//    struct tm ltime;
	//    char timestr[16];
	//    time_t local_tv_sec;

	/*
	* unused variables
	*/
	//    (VOID)(param);
	//    (VOID)(pkt_data);

	/* convert the timestamp to readable format */
	//    local_tv_sec = header->ts.tv_sec;
	//    localtime_s(&ltime, &local_tv_sec);
	//    strftime( timestr, sizeof timestr, "%H:%M:%S", &ltime);

	
	if (pkt_header->len<1000)return;
	if (((*(pkt_data + 36) << 8) | (*(pkt_data + 37))) != 5000)
	{
		//printf("\nport:%d",((*(pkt_data+36)<<8)|(*(pkt_data+37))));
		return;
	}
	
	u_char* data = (u_char*)pkt_data + UDP_HEADER_LEN;
	if (data[0] == 0)		//I chanel first part
	{
		iRec++;
		if (iRec >= MAX_IREC)iRec = 0;
		memcpy(dataBuff[iRec].header, data, FRAME_HEADER_SIZE);
		memcpy(dataBuff[iRec].dataI, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 2) //Q chanel first part
	{
		memcpy(dataBuff[iRec].dataQ, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 1) //I chanel second part
	{
		memcpy(dataBuff[iRec].dataI+1024, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 3) //Q chanel second part
	{
		memcpy(dataBuff[iRec].dataQ+1024, data + FRAME_HEADER_SIZE, 1024);
	}
	return;
	//    printf("len:%d\n", header->len);
	//    //printf("%.6d len:%d\n", header->ts.tv_usec, header->len);
	//    for(short i=0;i<dataB[iRec].len;i++)
	//    {
	//        printf("%x-",dataB[iRec].data[i]);
	//    }
	//    printf("\n");

}
