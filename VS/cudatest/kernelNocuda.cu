#include "hip/hip_runtime.h"
//setx -m OPENCV_DIR D:\OpenCV\OpenCV331\opencv\build
//setx path "%path%;D:\OpenCV\OpenCV331\opencv\build\bin\Release\"
//#include ""
//#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <winsock2.h>
#include <windows.h>
#include <conio.h>
#include <tchar.h>
#define HAVE_REMOTE// for pcap
#include "pcap.h"
#define HR2D_PK//
#define FRAME_LEN 1024
#define FFT_SIZE 1
#define BANG_KHONG 0
//int mFFTSkip = FFT_SIZE/8;
#include <time.h>
#define MAX_IREC 2400
#pragma comment(lib, "user32.lib")
#pragma comment (lib, "Ws2_32.lib")
//file mapping
#define FRAME_HEADER_SIZE 34
bool isUnsigned = false;
//using namespace cv;
using namespace std;

#include <stdio.h>

//_______________________________________________________________________

struct DataFrame// buffer for data frame
{
	char header[FRAME_HEADER_SIZE];
	char dataI[FRAME_LEN];
	char dataQ[FRAME_LEN];
	char dataUnsign[FRAME_LEN];
	char image256[256];
	bool isToFFT;
} dataBuff[MAX_IREC];

#define OUTPUT_FRAME_SIZE FRAME_LEN*2+FRAME_HEADER_SIZE

u_char outputFrame[OUTPUT_FRAME_SIZE];

int iProcessing=0,iReady = 50;
void packet_handler(u_char *param, const struct pcap_pkthdr *header, const u_char *pkt_data);
void pcapRun();


int mSocket;
struct sockaddr_in si_peter;
struct sockaddr_in si_capin;
void socketInit()
{
	WSADATA wsa;
	//Initialise winsock
	printf("\nInitialising Winsock...");
	if (WSAStartup(MAKEWORD(2, 2), &wsa) != 0)
	{
		printf("Failed. Error Code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}
	printf("Initialised.\n");
	//init socket for UDP connect to Peter
	mSocket = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP);
	
	if (mSocket == SOCKET_ERROR)
	{
		printf("socket() failed with error code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}
			
	memset((char *)&si_capin, 0, sizeof(si_capin));
	si_capin.sin_family = AF_INET;
	si_capin.sin_port = htons(34566);//port "127.0.0.1"
	si_capin.sin_addr.S_un.S_addr = inet_addr("127.0.0.1");
	int ret = bind(mSocket, (struct sockaddr *)&si_capin, sizeof(struct sockaddr));
	if (ret==-1)
	{
		printf("Port busy");
		exit(EXIT_FAILURE);
	}
	//setup address structure
	memset((char *)&si_peter, 0, sizeof(si_peter));
	si_peter.sin_family = AF_INET;
	si_peter.sin_port = htons(31000);//port "127.0.0.1"
	si_peter.sin_addr.S_un.S_addr = inet_addr("127.0.0.1");

}
void socketDelete()
{
	closesocket(mSocket);
	WSACleanup();
}

DWORD WINAPI ProcessDataBuffer(LPVOID lpParam);
DWORD WINAPI ProcessCommandBuffer(LPVOID lpParam);
void StartProcessing()
{
	CreateThread(
		NULL,                   // default security attributes
		0,                      // use default stack size  
		ProcessDataBuffer,       // thread function name
		NULL,          // argument to thread function 
		0,                      // use default creation flags 
		NULL);   // returns the thread identifier 
	CreateThread(
		NULL,                   // default security attributes
		0,                      // use default stack size  
		ProcessCommandBuffer,       // thread function name
		NULL,          // argument to thread function 
		0,                      // use default creation flags 
		NULL);   // returns the thread identifier 

}


FILE* pFile;
int main()
{

	/* start the capture */
	socketInit();

	StartProcessing();

	char fileName[50];
	time_t times = time(NULL);
	int strlen = sprintf(fileName, "raw_data_record_%d.dat", times);
	pFile = fopen(fileName, "wb");
	if (!pFile)
	{
		printf("\n Error creating .dat file");
	}
	else
	{
		printf("\nRecord start, file name:");
		printf(fileName);
	}
	
	pcapRun();
	fclose(pFile);
    return 0;
}
//precompiling code for FFT

// Complex data type


void pcapRun()
{
	pcap_if_t *alldevs;
	pcap_if_t *d;
	pcap_t *adhandle;
	char errbuf[PCAP_ERRBUF_SIZE];
	//
	/* Retrieve the device list on the local machine */
	if (pcap_findalldevs_ex((char*)PCAP_SRC_IF_STRING, NULL, &alldevs, errbuf) == -1)
	{
		//isRunning = false;
		printf(errbuf); return;
	}
	//isRunning = true;
	int i = 0;
	/* Print the list */
	for (d = alldevs; d; d = d->next)
	{
		printf("%d. %s", ++i, d->name);
		if (d->description)
			printf(" (%s)\n", d->description);
		else
			printf(" (No description available)\n");
	}
	d = alldevs;
	if ((adhandle = pcap_open(d->name,          // name of the device
		65536,            // portion of the packet to capture
		// 65536 guarantees that the whole packet will be captured on all the link layers
		PCAP_OPENFLAG_PROMISCUOUS,    // promiscuous mode
		1000,             // read timeout
		NULL,             // authentication on the remote machine
		errbuf            // error buffer
		)) == NULL)
	{
		/* Free the device list */
		pcap_freealldevs(alldevs);
		return;
	}
	printf("\nlistening on %s...\n", d->description);
	pcap_loop(adhandle, 0, packet_handler, NULL);
}
u_char dataOut[FRAME_LEN];
long int nFrames = 0;

float2 ramSignalTL[FRAME_LEN][FFT_SIZE];
float2 ramSignalNen[MAX_IREC][FRAME_LEN];
float2 ramImage[FRAME_LEN];
char recvDatagram[1000];
DWORD WINAPI ProcessCommandBuffer(LPVOID lpParam)
{
	while (false)
	{
		int PeterAddrSize = sizeof (si_peter);
		int iResult = recvfrom(mSocket, recvDatagram, 1000, 0, (struct sockaddr *) &si_peter, &PeterAddrSize);
		if (iResult == SOCKET_ERROR) {
			wprintf(L"recvfrom failed with error %d\n", WSAGetLastError());
		}
	}
	return 0;
}
/*
int datatestI[MAX_IREC];
int datatestQ[MAX_IREC];
int datatestA[MAX_IREC];*/

DWORD WINAPI ProcessDataBuffer(LPVOID lpParam)
{
	
	
	while (true)
	{
		Sleep(1);
		
		
		while (iProcessing!= iReady )
		{
			
			for (int ir = 0; ir < FRAME_LEN; ir++)
			{
				//ramSignalNen[iProcessing][ir].x = sqrt(double(dataBuff[iProcessing].dataI[ir])*(dataBuff[iProcessing].dataI[ir]) + double(dataBuff[iProcessing].dataQ[ir])*(dataBuff[iProcessing].dataQ[ir]));
				/*if (ir == 260)
				{
					datatestI[iProcessing] = float(dataBuff[iProcessing].dataI[264]);
					datatestQ[iProcessing] = float(dataBuff[iProcessing].dataQ[264]);
					datatestA[iProcessing] = sqrt(double(dataBuff[iProcessing].dataI[264] * dataBuff[iProcessing].dataI[264] + dataBuff[iProcessing].dataQ[264] * dataBuff[iProcessing].dataQ[264]));
					//ramSignalNen[iProcessing][ir].x = int(dataBuff[iProcessing].dataI[264]);
					//ramSignalNen[iProcessing][ir].y = int(dataBuff[iProcessing].dataQ[264]);
				}*/
				//ramSignalNen[iProcessing][ir].x = sqrt(double(dataBuff[iProcessing].dataI[ir] * dataBuff[iProcessing].dataI[ir] + dataBuff[iProcessing].dataQ[ir] * dataBuff[iProcessing].dataQ[ir]));//int(dataBuff[iProcessing].dataI[ir]);
				ramSignalNen[iProcessing][ir].x = float(dataBuff[iProcessing].dataI[ir]);
				ramSignalNen[iProcessing][ir].y = float(dataBuff[iProcessing].dataQ[ir]);//0;// 
				//ramSignalNen[iProcessing][ir].y = 0;
			}
			

			/*if (!dataBuff[iProcessing].isToFFT)
			{
				//jump to next period
				iProcessing++;
				if (iProcessing >= MAX_IREC)iProcessing = 0;
				continue;
			}*/
			int ia;
			for (int ir = 0; ir < FRAME_LEN; ir++)
			{
				ia = iProcessing;
				for (int i = 0; i < FFT_SIZE; i++)
				{
					ramSignalTL[ir][i] = ramSignalNen[ia][ir];
					ia--;
					if (ia < 0)ia += MAX_IREC;
				}
			}
			
			
			memcpy(outputFrame, dataBuff[iProcessing].header, FRAME_HEADER_SIZE);
			if(isUnsigned)
			{
				memcpy(outputFrame + FRAME_HEADER_SIZE, dataBuff[iProcessing].dataUnsign, FRAME_LEN);
			}
			else 
			for (int i = 0; i < FRAME_LEN; i++)
			{
				int maxAmp = 0, indexMaxFFT = 0;
				//for (int j = 0; j<FFT_SIZE; j++)
				int fftSkip = BANG_KHONG*FFT_SIZE / 16.0;
				for (int j = fftSkip; j<FFT_SIZE - fftSkip; j++)
				{
					int ampl = (ramSignalTL[i][j].x * ramSignalTL[i][j].x) + (ramSignalTL[i][j].y * ramSignalTL[i][j].y);
					if (ampl>maxAmp)
					{
						maxAmp = ampl;
						indexMaxFFT = j;
					}
				}
				/*if (i == 264)
				{
					datatestA[iProcessing] = u_char(sqrt(float(maxAmp)) / float(FFT_SIZE));//int(sqrt(float(maxAmp/16.0)));
				}*/
				//outputFrame[i + FRAME_HEADER_SIZE] = u_char(sqrt(float(maxAmp)));
				/*if (i%2)
				outputFrame[i + FRAME_HEADER_SIZE] = u_char(datatestI[i] + 60);
				else
				outputFrame[i + FRAME_HEADER_SIZE] = u_char(datatestQ[i]+60);//u_char(sqrt(float(maxAmp)) / float(1 ));*/
				int res = sqrt(float(maxAmp)) / float(FFT_SIZE);
				if (res > 255)res = 255;
				outputFrame[i + FRAME_HEADER_SIZE] = res;// u_char(sqrt(float(maxAmp)) / float(FFT_SIZE));
				outputFrame[i + FRAME_LEN + FRAME_HEADER_SIZE] = u_char(indexMaxFFT*16.0 / (FFT_SIZE));
			}
			sendto(mSocket, (char*)outputFrame, OUTPUT_FRAME_SIZE, 0, (struct sockaddr *) &si_peter, sizeof(si_peter));
			
			//jump to next period
			iProcessing++;
			if (iProcessing >= MAX_IREC)iProcessing = 0;
			
		}
	}
	


}

#define UDP_HEADER_LEN 42
void packet_handler(u_char *param, const struct pcap_pkthdr *pkt_header, const u_char *pkt_data)
{
	//    struct tm ltime;
	//    char timestr[16];
	//    time_t local_tv_sec;

	/*
	* unused variables
	*/
	//    (VOID)(param);
	//    (VOID)(pkt_data);

	/* convert the timestamp to readable format */
	//    local_tv_sec = header->ts.tv_sec;
	//    localtime_s(&ltime, &local_tv_sec);
	//    strftime( timestr, sizeof timestr, "%H:%M:%S", &ltime);

	//if (pkt_header->len<1000)return;
	
	//pkt_header->
	/*if (((*(pkt_data + 36) << 8) | (*(pkt_data + 37))) != 5000)
	{
		//printf("\nport:%d",((*(pkt_data+36)<<8)|(*(pkt_data+37))));
		return;
	}*/
	//int iNext = iReady + 1;
	//if (iNext >= MAX_IREC)iNext = 0;
	
	u_char* data = (u_char*)pkt_data + UDP_HEADER_LEN;
	unsigned char len1 = (pkt_header->len)>>8;
	unsigned char len2 = (pkt_header->len);
	fwrite(&len1, 1,1 , pFile);
	fwrite(&len2, 1, 1, pFile);
	fwrite(data, 1,pkt_header->len , pFile);
	if (ftell(pFile) > 100000000)
	{
		fclose(pFile);
		char fileName[50];
		time_t times = time(NULL);
		int strlen = sprintf(fileName, "raw_data_record_%d.dat", times);
		pFile = fopen(fileName, "wb");
		if (!pFile)
		{
			printf("\n Error creating .dat file");
		}
		else
		{
			printf("\nRecord start, file name:");
			printf(fileName);
		}
	}
	/*if (data[0] == 1)		//I chanel first part
	{
		isUnsigned = false;
		//dataBuff[iNext].isToFFT = ((iNext%mFFTSkip) == 0);
		memcpy(dataBuff[iNext].header, data, FRAME_HEADER_SIZE);
		memcpy(dataBuff[iNext].dataI, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 2) //Q chanel first part
	{
		memcpy(dataBuff[iNext].dataQ, data + FRAME_HEADER_SIZE, 1024);
		iReady++;
		if (iReady >= MAX_IREC)iReady = 0;
	}
	else if (data[0] == 3) //tin hieu gia lien tuc
	{
		isUnsigned = true;
		memcpy(dataBuff[iNext].header, data, FRAME_HEADER_SIZE);
		//memcpy(dataBuff[iNext].dataI, data + FRAME_HEADER_SIZE, 1024);
		//memset(dataBuff[iNext].dataQ, 0, 1024);
		memcpy(dataBuff[iNext].dataUnsign, data + FRAME_HEADER_SIZE, FRAME_LEN);
		iReady++;
		if (iReady >= MAX_IREC)iReady = 0;
	}
	return;*/
}
