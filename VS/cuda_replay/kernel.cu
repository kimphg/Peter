﻿//setx -m OPENCV_DIR D:\OpenCV\OpenCV331\opencv\build
//setx path "%path%;D:\OpenCV\OpenCV331\opencv\build\bin\Release\"
//#include ""
//#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <winsock2.h>
#include <windows.h>
#include <conio.h>
#include <tchar.h>
#define HAVE_REMOTE// for pcap
#include "pcap.h"
#define HR2D_PK//
#define FRAME_LEN 1024
#define FFT_SIZE 1
#define BANG_KHONG 0
//int mFFTSkip = FFT_SIZE/8;
#include <time.h>
#define MAX_IREC 2400
#pragma comment(lib, "user32.lib")
#pragma comment (lib, "Ws2_32.lib")
//file mapping
#define FRAME_HEADER_SIZE 34
bool isUnsigned = false;
//using namespace cv;
using namespace std;

#include <stdio.h>

//_______________________________________________________________________

struct DataFrame// buffer for data frame
{
	char header[FRAME_HEADER_SIZE];
	char dataI[FRAME_LEN];
	char dataQ[FRAME_LEN];
	char dataUnsign[FRAME_LEN];
	char image256[256];
	bool isToFFT;
} dataBuff[MAX_IREC];

#define OUTPUT_FRAME_SIZE FRAME_LEN*2+FRAME_HEADER_SIZE

u_char outputFrame[OUTPUT_FRAME_SIZE];

int iProcessing=0,iReady = 50;
void packet_handler(u_char *param, const struct pcap_pkthdr *header, const u_char *pkt_data);
void pcapRun();


int mSocket;
struct sockaddr_in si_peter;
struct sockaddr_in si_capin;
void socketInit()
{
	WSADATA wsa;
	//Initialise winsock
	printf("\nInitialising Winsock...");
	if (WSAStartup(MAKEWORD(2, 2), &wsa) != 0)
	{
		printf("Failed. Error Code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}
	printf("Initialised.\n");
	//init socket for UDP connect to Peter
	mSocket = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP);
	
	if (mSocket == SOCKET_ERROR)
	{
		printf("socket() failed with error code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}
			
	memset((char *)&si_capin, 0, sizeof(si_capin));
	si_capin.sin_family = AF_INET;
	si_capin.sin_port = htons(34565);//port "127.0.0.1"
	si_capin.sin_addr.S_un.S_addr = inet_addr("192.168.0.70");
	int ret = bind(mSocket, (struct sockaddr *)&si_capin, sizeof(struct sockaddr));
	if (ret==-1)
	{
		printf("Port busy");
		exit(EXIT_FAILURE);
	}
	//setup address structure
	memset((char *)&si_peter, 0, sizeof(si_peter));
	si_peter.sin_family = AF_INET;
	si_peter.sin_port = htons(5000);//port "127.0.0.1"
	si_peter.sin_addr.S_un.S_addr = inet_addr("192.168.0.55");

}
void socketDelete()
{
	closesocket(mSocket);
	WSACleanup();
}


FILE* pFile;
char buff[3000];
void ReplayData()
{
	unsigned char len1,len2;
	unsigned long long dataSize = 0;
	while (!feof(pFile))
	{

		fread(&len1, 1, 1, pFile);
		fread(&len2, 1, 1, pFile);
		int len = (len1 << 8) | len2;
		if (len > 3000)
		{
			printf("\nwrong datalen, total data sent:%d", dataSize);
			break;
		}
		fread(buff, 1, len, pFile);
		dataSize += len;
		 sendto(mSocket, buff, len, 0, (struct sockaddr *) &si_peter, sizeof(si_peter));
		Sleep(5);
	}
	printf("\n total data sent:%d", dataSize);
}

int main(int argc, char **argv)
{

	/* start the capture */
	socketInit();

	//StartProcessing();
	if (argc < 2)
	{
		printf("\nNo file selected");
		return;
	}
	char *fileName = argv[1];
	//int strlen = sprintf(fileName, "raw_data_record_%d.dat", times);
	pFile = fopen(fileName, "rb");
	if (!pFile)
	{
		printf("\n Error opening .dat file");
	}
	else
	{
		printf("\nReplay start, file name:");
		printf(fileName);
	}
	
	ReplayData();
	fclose(pFile);
	socketDelete();
    return 0;
}
//precompiling code for FFT

// Complex data type


/*
int datatestI[MAX_IREC];
int datatestQ[MAX_IREC];
int datatestA[MAX_IREC];*/
