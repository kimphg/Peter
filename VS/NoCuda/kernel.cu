#include "hip/hip_runtime.h"
﻿//setx -m OPENCV_DIR D:\OpenCV\OpenCV331\opencv\build
//setx path "%path%;D:\OpenCV\OpenCV331\opencv\build\bin\Release\"
#include ""
//#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <winsock2.h>
#include <windows.h>
#include <conio.h>
#include <tchar.h>
#define HAVE_REMOTE// for pcap
#include "pcap.h"
#define HR2D_PK//
#define FRAME_LEN 2048
#define OUTPUT_FRAME_SIZE FRAME_LEN*2+FRAME_HEADER_SIZE
#define FFT_SIZE 32
#define BANG_KHONG 0
int mFFTSkip = (FFT_SIZE/10);

#define MAX_IREC 2400
#pragma comment(lib, "user32.lib")
#pragma comment (lib, "Ws2_32.lib")
//file mapping
#define FRAME_HEADER_SIZE 34

//using namespace cv;
using namespace std;
//#include "FFTcore.cuh"
// includes for FFT
#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdio.h>

__global__ void complexMulKernel(hipfftComplex *res, const hipfftComplex *v1, const hipfftComplex *v2)
{
	int i = threadIdx.x;
	res[i].x = (v1[i].x * v2[i].x + v1[i].y * (v2[i].y)) ;
	res[i].y = (v1[i].x * (-v2[i].y) + v1[i].y * v2[i].x) ;
}
class coreFFT
{
public:

	bool isActive;
	hipfftHandle planTL;
	//hipfftHandle planNenTH;
	//hipfftHandle planImageFFT;
	hipfftComplex *dSignalTL;
	//hipfftComplex *dSignalNenRes;
	//hipfftComplex *dSignalNen;
	//hipfftComplex *dImageNen;
	int mMemSizeTL;
	int mMemSizeNen;
	int mMemSizeImage;
	int mTichLuySize;//16
	int mFrameLen;
	coreFFT(int frameLen, int ntichluy)
	{
		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			isActive = false;
			return;
		}
		else
		{
			printf("\ncudaSetDevice on ");
		}
		mFrameLen = frameLen;
		mTichLuySize = ntichluy;
		mMemSizeTL = sizeof(hipfftComplex)* mTichLuySize*frameLen;
		//mMemSizeNen = sizeof(hipfftComplex)*frameLen;
		//mMemSizeImage = sizeof(hipfftComplex)*frameLen;


		if (hipfftPlan1d(&planTL, mTichLuySize, HIPFFT_C2C, frameLen) != HIPFFT_SUCCESS)
		{
			printf("\nFFT planTL failed to init");
			isActive = false;
			return;
		}
		// Allocate device memory for signal tich luy
		hipMalloc((void **)&dSignalTL, mMemSizeTL);
		/*
		if (hipfftPlan1d(&planNenTH, frameLen, HIPFFT_C2C, 1) != HIPFFT_SUCCESS)printf("\nFFT planNenTH failed to init");
		// Allocate device memory for signal nen
		hipMalloc((void **)&dSignalNen, mMemSizeNen);
		hipMalloc((void **)&dSignalNenRes, mMemSizeNen);
		*/
		//if (hipfftPlan1d(&planImageFFT, frameLen, HIPFFT_C2C, 1) != HIPFFT_SUCCESS)printf("\nFFT planTL failed to init");
		// Allocate device memory for image nen
		//hipMalloc((void **)&dImageNen, mMemSizeNen);
	}
	void exeFFTTL(hipfftComplex *h_signal)
	{
		hipMemcpy(dSignalTL, h_signal, mMemSizeTL, hipMemcpyHostToDevice);
		hipfftExecC2C(planTL, dSignalTL, dSignalTL, HIPFFT_FORWARD);
		
		if (hipGetLastError() != hipSuccess) {
			fprintf(stderr, "FFT kernel launch failed: %s\n", hipGetErrorString(hipGetLastError()));
			return;
		}
		hipMemcpy(h_signal, dSignalTL, mMemSizeTL, hipMemcpyDeviceToHost);
	}
	/*void exeFFTNen(hipfftComplex *h_signal, hipfftComplex* h_image)
	{
		//move signal to gpu and process fft forward
		hipMemcpy(dSignalNen, h_signal, mMemSizeNen, hipMemcpyHostToDevice);
		hipfftExecC2C(planNenTH, dSignalNen, dSignalNen, HIPFFT_FORWARD);
		//move image to gpu and process fft forward
		hipMemcpy(dImageNen, h_image, mMemSizeNen, hipMemcpyHostToDevice);
		hipfftExecC2C(planNenTH, dImageNen, dImageNen, HIPFFT_FORWARD);
		// Element wise complext multiplication
		for (int i = 0; i < mFrameLen; i += 1024)
		{
			complexMulKernel << <1, 1024 >> >(dSignalNenRes+i, dSignalNen+i, dImageNen+i);
		}
		
		hipfftExecC2C(planNenTH, dSignalNenRes, dSignalNenRes, HIPFFT_FORWARD);
		
		hipMemcpy(h_signal, dSignalNenRes, mMemSizeNen, hipMemcpyDeviceToHost);
	}*/
	~coreFFT()
	{
		hipfftDestroy(planTL);
		//hipfftDestroy(planNenTH);
		// cleanup memory
		hipFree(dSignalTL);
		//hipFree(dSignalNen);
		//hipFree(dImageNen);
	}
};
//_______________________________________________________________________

struct DataFrame// buffer for data frame
{
	char header[FRAME_HEADER_SIZE];
	char dataPM_I[FRAME_LEN];
	char dataPM_Q[FRAME_LEN];
	short dataLen;
	bool isToFFT;
} dataBuff[MAX_IREC];
unsigned int gyroValue = 0;

u_char outputFrame[OUTPUT_FRAME_SIZE];

int iProcessing=0,iReady = 50;
void packet_handler(u_char *param, const struct pcap_pkthdr *header, const u_char *pkt_data);
void pcapRun();


int mSocket;
struct sockaddr_in si_peter;
struct sockaddr_in si_capin;
void socketInit()
{
	WSADATA wsa;
	//Initialise winsock
	printf("\nInitialising Winsock...");
	if (WSAStartup(MAKEWORD(2, 2), &wsa) != 0)
	{
		printf("Failed. Error Code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}
	printf("Initialised.\n");
	//init socket for UDP connect to Peter
	mSocket = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP);
	
	if (mSocket == SOCKET_ERROR)
	{
		printf("socket() failed with error code : %d", WSAGetLastError());
		exit(EXIT_FAILURE);
	}
			
	memset((char *)&si_capin, 0, sizeof(si_capin));
	si_capin.sin_family = AF_INET;
	si_capin.sin_port = htons(34566);//port "127.0.0.1"
	si_capin.sin_addr.S_un.S_addr = inet_addr("127.0.0.1");
	int ret = bind(mSocket, (struct sockaddr *)&si_capin, sizeof(struct sockaddr));
	if (ret==-1)
	{
		printf("Port busy");
		exit(EXIT_FAILURE);
	}
	//setup address structure
	memset((char *)&si_peter, 0, sizeof(si_peter));
	si_peter.sin_family = AF_INET;
	si_peter.sin_port = htons(31000);//port "127.0.0.1"
	si_peter.sin_addr.S_un.S_addr = inet_addr("127.0.0.1");

}
void socketDelete()
{
	closesocket(mSocket);
	WSACleanup();
}

DWORD WINAPI ProcessDataBuffer(LPVOID lpParam);
DWORD WINAPI ProcessCommandBuffer(LPVOID lpParam);
void StartProcessing()
{
	CreateThread(
		NULL,                   // default security attributes
		0,                      // use default stack size  
		ProcessDataBuffer,       // thread function name
		NULL,          // argument to thread function 
		0,                      // use default creation flags 
		NULL);   // returns the thread identifier 
	CreateThread(
		NULL,                   // default security attributes
		0,                      // use default stack size  
		ProcessCommandBuffer,       // thread function name
		NULL,          // argument to thread function 
		0,                      // use default creation flags 
		NULL);   // returns the thread identifier 

}
coreFFT *mFFT;


int main()
{

	/* start the capture */
	socketInit();
	mFFT = new coreFFT(FRAME_LEN, FFT_SIZE);
	StartProcessing();
	pcapRun();
	
    return 0;
}
//precompiling code for FFT

// Complex data type


void pcapRun()
{
	pcap_if_t *alldevs;
	pcap_if_t *d;
	pcap_t *adhandle;
	char errbuf[PCAP_ERRBUF_SIZE];
	//
	/* Retrieve the device list on the local machine */
	if (pcap_findalldevs_ex((char*)PCAP_SRC_IF_STRING, NULL, &alldevs, errbuf) == -1)
	{
		//isRunning = false;
		printf(errbuf); return;
	}
	//isRunning = true;
	int i = 0;
	/* Print the list */
	for (d = alldevs; d; d = d->next)
	{
		printf("\n%d. %s", ++i, d->name);
		if (d->description)
			printf(" (%s)\n", d->description);
		else
			printf(" (No description available)\n");
	}
	d = alldevs;
	if ((adhandle = pcap_open(d->name,          // name of the device
		65536,            // portion of the packet to capture
		// 65536 guarantees that the whole packet will be captured on all the link layers
		PCAP_OPENFLAG_PROMISCUOUS,    // promiscuous mode
		1000,             // read timeout
		NULL,             // authentication on the remote machine
		errbuf            // error buffer
		)) == NULL)
	{
		/* Free the device list */
		pcap_freealldevs(alldevs);
		return;
	}
	printf("\nlistening on %s...\n", d->description);
	pcap_loop(adhandle, 0, packet_handler, NULL);
}
u_char dataOut[FRAME_LEN];
long int nFrames = 0;

hipfftComplex ramSignalTL[FRAME_LEN][FFT_SIZE];
hipfftComplex ramSignalNen[MAX_IREC][FRAME_LEN];
hipfftComplex ramImage[FRAME_LEN];
char recvDatagram[1000];
DWORD WINAPI ProcessCommandBuffer(LPVOID lpParam)
{
	while (false)
	{
		int PeterAddrSize = sizeof (si_peter);
		int iResult = recvfrom(mSocket, recvDatagram, 1000, 0, (struct sockaddr *) &si_peter, &PeterAddrSize);
		if (iResult == SOCKET_ERROR) {
			wprintf(L"recvfrom failed with error %d\n", WSAGetLastError());
		}
	}
	return 0;
}
/*
int datatestI[MAX_IREC];
int datatestQ[MAX_IREC];
int datatestA[MAX_IREC];*/

DWORD WINAPI ProcessDataBuffer(LPVOID lpParam)
{
	
	
	while (true)
	{
		Sleep(1);
		
		
		while (iProcessing!= iReady )
		{
			
			for (int ir = 0; ir < FRAME_LEN; ir++)
			{
				
				//ramSignalNen[iProcessing][ir].x = sqrt(double(dataBuff[iProcessing].dataPM_I[ir] * dataBuff[iProcessing].dataPM_I[ir] + dataBuff[iProcessing].dataPM_Q[ir] * dataBuff[iProcessing].dataPM_Q[ir]));//int(dataBuff[iProcessing].dataPM_I[ir]);
				ramSignalNen[iProcessing][ir].x = float(dataBuff[iProcessing].dataPM_I[ir]);
				ramSignalNen[iProcessing][ir].y = float(dataBuff[iProcessing].dataPM_Q[ir]);//0;// 
				//ramSignalNen[iProcessing][ir].y = 0;
			}
			/*
			for (int ir = 0; ir < FRAME_LEN; ir++)
			{
				if (ir < 256)
				{
					ramImage[ir].x = ramSignalNen[iProcessing][ir+500].x;//dataBuff[iProcessing].image256[ir];
				}
				else
				{
					ramImage[ir].x = 0;
				}
				ramImage[ir].y = 0;
			}*/
			//bat dau loc nen anh guong
			//mFFT->exeFFTNen(ramSignalNen[iProcessing], ramImage);
			/*int sum = 0;
			for (int i = 0; i < FRAME_LEN; i++)
			{
				int outx =  int(sqrt((ramSignalNen[iProcessing][i].x)*(ramSignalNen[iProcessing][i].x) + (ramSignalNen[iProcessing][i].y)*(ramSignalNen[iProcessing][i].y)));
				sum += outx;
				if (outx>255)
					outx = 255;
				//bien bo
				outputFrame[i + FRAME_HEADER_SIZE] = outx;// u_char((ramSignalNen[iProcessing][i].x)*(ramSignalNen[iProcessing][i].x) + (ramSignalNen[iProcessing][i].y)*(ramSignalNen[iProcessing][i].y));
				//kenh dopler
				outputFrame[i + FRAME_LEN + FRAME_HEADER_SIZE] = 0;
			}
			if (sum / 2048.0 > 200)
			{
				sum = sum;
			}*/
			//memcpy((char*)outputFrame + FRAME_HEADER_SIZE, ramSignalNen[iProcessing],1024);
			//memcpy((char*)outputFrame + FRAME_HEADER_SIZE + 1024, ramSignalNen[iProcessing]+1024, 1024);
			//memcpy(outputFrame, dataBuff[iProcessing].header, FRAME_HEADER_SIZE);
			//sendto(mSocket, (char*)outputFrame, OUTPUT_FRAME_SIZE, 0, (struct sockaddr *) &si_peter, sizeof(si_peter));
			//
			//continue;
			//tich luy fft
			//if (!dataBuff[iProcessing].isToFFT)continue;

			//if (iProcessing >= MAX_IREC)iProcessing -= MAX_IREC;
			//nFrames++;

			if (!dataBuff[iProcessing].isToFFT)
			{
				//jump to next period
				iProcessing++;
				if (iProcessing >= MAX_IREC)iProcessing = 0;
				continue;
			}
			int ia;
			for (int ir = 0; ir < FRAME_LEN; ir++)
			{
				ia = iProcessing;
				for (int i = 0; i < FFT_SIZE; i++)
				{
					ramSignalTL[ir][i] = ramSignalNen[ia][ir];
					ia--;
					if (ia < 0)ia += MAX_IREC;
				}
			}
			
			/*if ((dataBuff[iProcessing].azi != oldAzi + 1) && (oldAzi!=2047))
			{
				printf("\nAzi:%d Count:%d", dataBuff[iProcessing].azi, nFrames);
				printf("Azi old:%d", oldAzi);
			}*/
			
			/*oldAzi = dataBuff[iProcessing].azi;
			*/
			//printf("\nAzi:%d Count:%d", dataBuff[iProcessing].azi, nFrames);
			
			// perform fft

			if (mFFT->isActive)mFFT->exeFFTTL((hipfftComplex*)ramSignalTL);
			dataBuff[iProcessing].header[32] = gyroValue >> 8;
			dataBuff[iProcessing].header[33] = gyroValue;
			
			memcpy(outputFrame, dataBuff[iProcessing].header, FRAME_HEADER_SIZE);
			
			for (int i = 0; i < FRAME_LEN; i++)
			{
				float maxAmp = 0;
				int indexMaxFFT = 0;
				//for (int j = 0; j<FFT_SIZE; j++)
				int fftSkip = BANG_KHONG*FFT_SIZE / 16;
				for (int j = fftSkip; j<FFT_SIZE - fftSkip; j++)
				{
					float ampl = (ramSignalTL[i][j].x * ramSignalTL[i][j].x) + (ramSignalTL[i][j].y * ramSignalTL[i][j].y);
					if (ampl>maxAmp)
					{
						maxAmp = ampl;
						indexMaxFFT = j;
					}
				}
				float res = sqrt(float(maxAmp) / float(FFT_SIZE));
				if (res > 255)res = 255;
				outputFrame[i + FRAME_HEADER_SIZE] = res;// u_char(sqrt(float(maxAmp)) / float(FFT_SIZE));
				outputFrame[i + FRAME_LEN + FRAME_HEADER_SIZE] = u_char(indexMaxFFT*16.0 / (FFT_SIZE));
			}
			sendto(mSocket, (char*)outputFrame, OUTPUT_FRAME_SIZE, 0, (struct sockaddr *) &si_peter, sizeof(si_peter));
			
			//jump to next period
			iProcessing++;
			if (iProcessing >= MAX_IREC)iProcessing = 0;
			
		}
	}
	


}

#define UDP_HEADER_LEN 42
void packet_handler(u_char *param, const struct pcap_pkthdr *pkt_header, const u_char *pkt_data)
{
	//    struct tm ltime;
	//    char timestr[16];
	//    time_t local_tv_sec;

	/*
	* unused variables
	*/
	//    (VOID)(param);
	//    (VOID)(pkt_data);

	/* convert the timestamp to readable format */
	//    local_tv_sec = header->ts.tv_sec;
	//    localtime_s(&ltime, &local_tv_sec);
	//    strftime( timestr, sizeof timestr, "%H:%M:%S", &ltime);

	if (pkt_header->len<200)return;
	int port = ((*(pkt_data + 36) << 8) | (*(pkt_data + 37)));
	if (port == 5000)
	{
		/*
		+ 0: 1024 byte đầu kênh I
		+ 1: 1024 byte sau kênh I
		+ 2: 1024 byte đầu kênh Q
		+ 3: 1024 byte sau kênh Q
		+ 4: 256 byte máy hỏi  
		+ 5: 1024 byte tín hiệu giả L/tục (512 byte đầu là I, 512 byte sau là Q)   
		+ 6: 1024 byte sau kênh I tín hiệu xung đơn  
		+ 7: 1024 byte sau kênh Q tín hiệu xung đơn 

		*/
		u_char* data = (u_char*)pkt_data + UDP_HEADER_LEN;
		int iNext = iReady + 1;
		if (iNext >= MAX_IREC)iNext = 0;
		memcpy(dataBuff[iNext].header, data, FRAME_HEADER_SIZE);
		bool isLastFrame = false;
		if (data[0] == 0)		//0: 1024 byte đầu kênh I
		{
			//memcpy(dataBuff[iNext].header, data, FRAME_HEADER_SIZE);
			memcpy(dataBuff[iNext].dataPM_I, data + FRAME_HEADER_SIZE, 1024);
		}
		else if (data[0] == 1) //1: 1024 byte sau kênh I
		{
			memcpy(dataBuff[iNext].dataPM_I + 1024, data + FRAME_HEADER_SIZE, 1024);
			
		}
		else if (data[0] == 2) //2: 1024 byte đầu kênh Q
		{
			memcpy(dataBuff[iNext].dataPM_Q, data + FRAME_HEADER_SIZE, 1024);
			
		}
		else if (data[0] == 3) //3: 1024 byte sau kênh Q
		{
			memcpy(dataBuff[iNext].dataPM_Q + 1024, data + FRAME_HEADER_SIZE, 1024);
			dataBuff[iNext].dataLen = FRAME_LEN;
			isLastFrame = true;
			
		}
		else if (data[0] == 5) //5: 1024 byte tín hiệu giả L/tục (512 byte đầu là I, 512 byte sau là Q) 
		{
			memcpy(dataBuff[iNext].dataPM_I, data + FRAME_HEADER_SIZE, 512);
			memcpy(dataBuff[iNext].dataPM_Q, data + FRAME_HEADER_SIZE + 512, 512);
			dataBuff[iNext].dataLen = 512;
			isLastFrame = true;
		}
		else if (data[0] == 6) //6: 1024 byte sau kênh I tín hiệu xung đơn 
		{
			memcpy(dataBuff[iNext].dataPM_I, data + FRAME_HEADER_SIZE, 1024);
			dataBuff[iNext].dataLen = 1024;
			
		}
		else if (data[0] == 7) //7: 1024 byte sau kênh Q tín hiệu xung đơn 
		{
			memcpy(dataBuff[iNext].dataPM_Q, data + FRAME_HEADER_SIZE,1024);
			dataBuff[iNext].dataLen = 1024;
			isLastFrame = true;
			
		}
		if (isLastFrame)
		{
			iReady++;
			dataBuff[iNext].isToFFT = ((iNext%mFFTSkip) == 0);
			if (iReady >= MAX_IREC)iReady = 0;
		}
		return;
	}
	
	
}
/*void packet_handler_compress(u_char *param, const struct pcap_pkthdr *pkt_header, const u_char *pkt_data)
{
	if (pkt_header->len<1000)return;
	if (((*(pkt_data + 36) << 8) | (*(pkt_data + 37))) != 5000)
	{
		return;
	}
	u_char* data = (u_char*)pkt_data + UDP_HEADER_LEN;
	if (data[0] == 0)		//I chanel first part
	{
		iReady++;
		if (iReady >= MAX_IREC)iReady = 0;
		memcpy(dataBuff[iReady].header, data, FRAME_HEADER_SIZE);
		memcpy(dataBuff[iReady].dataPM_I, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 2) //Q chanel first part
	{
		memcpy(dataBuff[iReady].dataPM_Q, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 1) //I chanel second part
	{
		memcpy(dataBuff[iReady].dataPM_I + 1024, data + FRAME_HEADER_SIZE, 1024);
	}
	else if (data[0] == 3) //Q chanel second part
	{
		memcpy(dataBuff[iReady].dataPM_Q + 1024, data + FRAME_HEADER_SIZE, 1024);
	}
	return;
}*/